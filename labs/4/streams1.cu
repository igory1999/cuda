#include <iostream>
#include <random>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <string>

using namespace std;

void init_h_array(float *a, int n)
{
  std::default_random_engine generator;
  std::uniform_real_distribution<float> distribution(-1.0,1.0);

  for(int i = 0; i < n; ++i)
    a[i] = distribution(generator);
} 

__global__ void map(float *d_a, int n, int m)
{
  int id = blockDim.x * blockIdx.x  + threadIdx.x;
  
  if(id < n)
    for(int i = 0; i < m; ++i)
      {
	d_a[id] = sinf(d_a[id]) + cosf(d_a[id]);
	d_a[id] = expf(d_a[id]);
	d_a[id] = rsqrtf(d_a[id]) - d_a[id];
      }
}

void print_sample(float *a, int m, string msg)
{
  cout << msg << endl;
  for(int i = 0; i < m; ++i)
    cout << "a[" << i << "] = " << a[i] << ", ";
  cout << endl;
}

int main()
{
  const int N = 1024*1024*128;
  const int threads_in_block = 1024;
  const int nstreams = 2;
  const size_t size = N*sizeof(float);
  const int batch = N/nstreams;
  const size_t batch_size = batch*sizeof(float);
  const dim3 block_size = dim3(threads_in_block, 1, 1);
  const dim3 grid_size = dim3(N/block_size.x, 1, 1);
  const int iterations = 15;

  float *h_a = NULL;
  hipHostMalloc(&h_a, size, hipHostMallocDefault);
  init_h_array(h_a, N);

  print_sample(h_a, 5, "initial array");

  hipStream_t stream[nstreams];
  for(int i = 0; i < nstreams; ++i)
    hipStreamCreate(&stream[i]);

  float *d_a = NULL;
  hipMalloc((void **)&d_a, size);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for(int i = 0; i < nstreams; ++i)
    {
      hipMemcpyAsync(d_a + i*batch, h_a + i*batch, batch_size, hipMemcpyHostToDevice, stream[i]);
      map<<<grid_size, block_size, 0, stream[i]>>>(d_a + i*batch, batch, iterations);
      hipMemcpyAsync(h_a + i*batch, d_a + i*batch, batch_size, hipMemcpyDeviceToHost, stream[i]);
    }

  hipEventRecord(stop); 
  hipEventSynchronize(stop);
  float milliseconds = 0;  
  hipEventElapsedTime(&milliseconds, start, stop);

  print_sample(h_a, 5, "final array");

  cout << "H2D, kernel, D2H took " << milliseconds << " ms" << endl;
  
  hipFree(d_a);
  hipHostFree(h_a);
}
