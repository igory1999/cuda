
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>

using namespace std;



struct random_d_array
{
  float *data;
  int n;

  random_d_array(int n) :n{n}
  {
    hipMalloc((void**)&data, n*sizeof(float));
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandGenerateUniform(gen, data, n);
  }
  
  ~random_d_array()
  {
    hipFree(&data);
  }
};

__global__ void MyKernel(float *d, float *a, float *b, int n)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < n)
    d[idx] = a[idx] * b[idx];
}

int main()
{
  int numBlocks;
  int blockSize = 32;
  
  int device;
  hipDeviceProp_t prop;
  int activeWarps;
  int maxWarps;

  int N = 1024*1024;
  random_d_array a(N);
  random_d_array b(N);
  random_d_array d(N);

  string buffer;
  while(true)
    {
      cout << "Enter the block size or q to exit" << endl;
      cin >> buffer;
      if(buffer == "q") break;
      blockSize = stoi(buffer, nullptr);

      int gridSize = (N + blockSize - 1)/blockSize;

      cout << "blockSize = " << blockSize << ", gridSize = " << gridSize << endl;

      hipGetDevice(&device);
      hipGetDeviceProperties(&prop, device);
      
      hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, MyKernel, blockSize, 0);
      
      activeWarps = numBlocks * blockSize/prop.warpSize;
      maxWarps = prop.maxThreadsPerMultiProcessor/prop.warpSize;

      cout << "Occupancy: " << (double)activeWarps/maxWarps * 100 << "%" << endl;

      double average = 0.0;
      int iterations = 5;
      
      for(int i = 0; i < iterations; ++i)
	{
	  hipEvent_t start, stop;
	  hipEventCreate(&start);
	  hipEventCreate(&stop);
	  hipEventRecord(start);
	  
	  MyKernel<<<gridSize,blockSize>>>(d.data, a.data, b.data, N);
	  
	  hipEventRecord(stop); 
	  hipEventSynchronize(stop);
	  float milliseconds = 0;  
	  hipEventElapsedTime(&milliseconds, start, stop);
	  hipEventDestroy(start);
	  hipEventDestroy(stop);
	  cout << milliseconds << " ms" << endl;
	  if(i > 0)
	    average += milliseconds;
	}
      average /= (iterations - 1);
      cout << "Average = " << average << endl;
    }
}
